#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <string>
#include <queue>
#include <algorithm>
#include <math.h>
#include <unistd.h>
#include <vector>
#include "../include/preprocessing.h"
#include "../include/common.h"

void show_level(ptr_graph ret, int topo_max)
{
    //int topo_max = 2;
    queue<ptr_node> tmp_queue;
    for (int i = 0; i < ret->start_nodes.size(); i++)
    {
        tmp_queue.push(ret->start_nodes[i]);
    }
    while (!tmp_queue.empty())
    {
        ptr_node tmp_node = tmp_queue.front();
        if (tmp_node->topo_level < topo_max)
        {
            for (auto i = tmp_node->child.begin(); i != tmp_node->child.end(); i++)
            {
                (*i)->in_degree_tmp++;
                if ((*i)->in_degree_tmp == (*i)->in_degree)
                {
                    tmp_queue.push(*i);
                    (*i)->in_degree_tmp = 0;
                }
            }
        }
        printf("node id %d row start %d topo_level %d\n", tmp_node->id, tmp_node->info.start_row, tmp_node->topo_level);
        tmp_queue.pop();
    }
    printf("\n");
}

// Generate dependency graph every (row_block) rows
ptr_graph generate_graph_row_block(const int m, const int nnz,
                const int *csrRowPtr, const int *csrColIdx,
                const int row_block)
{
    ptr_graph ret = new graph();

    ptr_node tmp_nodes[m / row_block];

    ptr_node previous_node = NULL;

    int count = 0;
    int count2 = 0;

    for (int row = 0; row < m; row += row_block)
    {
        int row_ed = min(row + row_block, m);
        int cur_row_block = row / row_block;

        int using_shared = 1;
        for (int row2 = row; row2 < row_ed; row2++)
            if (csrRowPtr[row2 + 1] - csrRowPtr[row2] > BUF_SIZE) using_shared = 0;

        if (using_shared) count2++;
        count++;

        tmp_nodes[cur_row_block] = new node(ret->global_node,
        row, row_ed, csrRowPtr[row_ed] - csrRowPtr[row], using_shared);

        ret->global_node++;

        vector<int> dep_list;

        int only_diag = 1;
        for (int idx = csrRowPtr[row]; idx < csrRowPtr[row_ed]; idx++)
        {
            int col_idx = csrColIdx[idx];
            int dep_row_block = col_idx / row_block;
            if (dep_row_block < cur_row_block)
            {
                dep_list.push_back(dep_row_block);
                only_diag = 0;
            }
        }

        sort(dep_list.begin(), dep_list.end());
        auto end_pos = unique(dep_list.begin(), dep_list.end());

        for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
        {
            int dep_row_block = *(dep_i);

            tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
            tmp_nodes[dep_row_block]->out_degree++;
            tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
            tmp_nodes[cur_row_block]->in_degree++;
            if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
                tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
            ret->global_edge++;
        }

        if (previous_node)
        {
            previous_node->locality_node = tmp_nodes[cur_row_block];
        }
        previous_node = tmp_nodes[cur_row_block];

        // Identify row blocks with only diagonal elements
        if (only_diag)
        {
            ret->start_nodes.push_back(tmp_nodes[cur_row_block]);
        }
    }

    printf("node_count %d count %d count2 %d\n", m / row_block, count, count2);

    return ret;
}

ptr_graph generate_graph_row_block_thresh(const int m, const int nnz,
                const int *csrRowPtr, const int *csrColIdx, int thresh)
{
    ptr_graph ret = new graph();
    
    ptr_node tmp_nodes[m];

    int hash_idx[m];

    for (int i = 0; i < m; i++) hash_idx[i] = -1;

    int row = 0;
    int node_count = 0;

    ptr_node previous_node = NULL;

    while (row < m)
    {
        int row_st = row;
        hash_idx[row] = node_count;
        if (csrRowPtr[row + 1] - csrRowPtr[row] > thresh)
        {
            hash_idx[row] = node_count;
            row++;
        }
        else
        {
            //while (row < m && csrRowPtr[row + 1] - csrRowPtr[row] <= thresh)
            while (row < m && row - row_st < WARP_SIZE && csrRowPtr[row + 1] - csrRowPtr[row] <= thresh)
            {
                hash_idx[row] = node_count;
                row++;
            }
        }

        int using_shared = 1;
        for (int row2 = row_st; row2 < row; row2++)
            if (csrRowPtr[row2 + 1] - csrRowPtr[row2] > BUF_SIZE) using_shared = 0;

        tmp_nodes[node_count] = new node(ret->global_node,
        row_st, row, csrRowPtr[row] - csrRowPtr[row_st], using_shared);
        ret->global_node++;

        int cur_row_block = hash_idx[row_st];

        vector<int> dep_list;

        int only_diag = 1;
        for (int idx = csrRowPtr[row_st]; idx < csrRowPtr[row]; idx++)
        {
            int col_idx = csrColIdx[idx];
            int dep_row_block = hash_idx[col_idx];
            if (dep_row_block < cur_row_block)
            {
                dep_list.push_back(dep_row_block);
                only_diag = 0;
            }
        }

        sort(dep_list.begin(), dep_list.end());
        auto end_pos = unique(dep_list.begin(), dep_list.end());

        for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
        {
            int dep_row_block = *(dep_i);

            tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
            tmp_nodes[dep_row_block]->out_degree++;
            tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
            tmp_nodes[cur_row_block]->in_degree++;
            if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
                tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
            ret->global_edge++;
        }

        if (previous_node)
        {
            previous_node->locality_node = tmp_nodes[cur_row_block];
        }
        previous_node = tmp_nodes[cur_row_block];

        // Identify row blocks with only diagonal elements
        if (only_diag)
        {
            ret->start_nodes.push_back(tmp_nodes[node_count]);
        }

        node_count++;

    }

    // for (auto i = tmp_nodes[0]->child.begin(); i != tmp_nodes[0]->child.end(); i++)
    //     printf("child id %d\n", (*i)->id);

    return ret;
}

ptr_graph generate_graph_row_block_nodep(const int m, const int nnz,
                const int *csrRowPtr, const int *csrColIdx, int thresh)
{
    ptr_graph ret = new graph();
    
    ptr_node tmp_nodes[m];

    int hash_idx[m];

    for (int i = 0; i < m; i++) hash_idx[i] = -1;

    int row = 0;
    int node_count = 0;

    ptr_node previous_node = NULL;

    while (row < m)
    {
        int row_st = row;
        hash_idx[row] = node_count;

        row++;
        int flag = 0;
        //if (node_count <= 10) printf("row_st %d\n", row_st);
        while (row < m && row - row_st < thresh)
        {
            //if (node_count <= 10) printf("row %d:", row);
            for (int i = csrRowPtr[row]; i < csrRowPtr[row + 1]; i++)
            {
                //if (node_count <= 10) printf(" %d", csrColIdx[i]);
                if (csrColIdx[i] >= row_st && csrColIdx[i] < row)
                {
                    flag = 1;
                    break;
                } 
            }
            //if (node_count <= 10) printf("\n");
            if (flag) break;
            hash_idx[row] = node_count;
            row++;
        }

        int using_shared = 1;
        for (int row2 = row_st; row2 < row; row2++)
            if (csrRowPtr[row] - csrRowPtr[row_st] > BUF_SIZE) using_shared = 0;

        tmp_nodes[node_count] = new node(ret->global_node,
        row_st, row, csrRowPtr[row] - csrRowPtr[row_st], using_shared);
        ret->global_node++;

        //if (node_count <= 10) printf("block size: %d\n", row - row_st);

        int cur_row_block = hash_idx[row_st];

        vector<int> dep_list;

        int only_diag = 1;
        for (int idx = csrRowPtr[row_st]; idx < csrRowPtr[row]; idx++)
        {
            int col_idx = csrColIdx[idx];
            int dep_row_block = hash_idx[col_idx];
            if (dep_row_block < cur_row_block)
            {
                dep_list.push_back(dep_row_block);
                only_diag = 0;
            }
        }

        sort(dep_list.begin(), dep_list.end());
        auto end_pos = unique(dep_list.begin(), dep_list.end());

        for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
        {
            int dep_row_block = *(dep_i);

            tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
            tmp_nodes[dep_row_block]->out_degree++;
            tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
            tmp_nodes[cur_row_block]->in_degree++;
            if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
                tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
            ret->global_edge++;
        }

        if (previous_node)
        {
            previous_node->locality_node = tmp_nodes[cur_row_block];
        }
        previous_node = tmp_nodes[cur_row_block];

        // Identify row blocks with only diagonal elements
        if (only_diag)
        {
            ret->start_nodes.push_back(tmp_nodes[node_count]);
        }

        node_count++;

    }

    //printf("Node count: %d\n", node_count);
    // for (auto i = tmp_nodes[0]->child.begin(); i != tmp_nodes[0]->child.end(); i++)
    //     printf("child id %d\n", (*i)->id);

    return ret;    
}

ptr_graph generate_graph_row_block_avg(const int m, const int nnz,
                const int *csrRowPtr, const int *csrColIdx, int thresh)
{
    ptr_graph ret = new graph();
    
    ptr_node tmp_nodes[m];

    int hash_idx[m];

    for (int i = 0; i < m; i++) hash_idx[i] = -1;

    ptr_node previous_node = NULL;

    for (int row_st = 0; row_st < m; row_st += WARP_SIZE)
    {
        int row_ed = min(row_st + WARP_SIZE, m);

        float avg_nnz = csrRowPtr[row_ed] - csrRowPtr[row_st];
        if (row_ed - row_st) avg_nnz /= (row_ed - row_st);

        if (avg_nnz < thresh)
        {
            //printf("thread_level\n");
            for (int i = row_st; i < row_ed; i++)
                hash_idx[i] = ret->global_node;
            
            int cur_row_block = ret->global_node;

            int using_shared = 1;
            for (int row2 = row_st; row2 < row_ed; row2++)
                if (csrRowPtr[row_ed] - csrRowPtr[row_st] > BUF_SIZE) using_shared = 0;

            tmp_nodes[cur_row_block] = new node(ret->global_node,
            row_st, row_ed, csrRowPtr[row_ed] - csrRowPtr[row_st], using_shared);
            ret->global_node++;

            vector<int> dep_list;

            int only_diag = 1;
            for (int idx = csrRowPtr[row_st]; idx < csrRowPtr[row_ed]; idx++)
            {
                int col_idx = csrColIdx[idx];
                int dep_row_block = hash_idx[col_idx];
                if (dep_row_block < cur_row_block)
                {
                    dep_list.push_back(dep_row_block);
                    only_diag = 0;
                }
            }

            sort(dep_list.begin(), dep_list.end());
            auto end_pos = unique(dep_list.begin(), dep_list.end());

            for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
            {
                int dep_row_block = *(dep_i);

                tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
                tmp_nodes[dep_row_block]->out_degree++;
                tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
                tmp_nodes[cur_row_block]->in_degree++;
                if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
                    tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
                ret->global_edge++;
            }

            if (previous_node)
            {
                previous_node->locality_node = tmp_nodes[cur_row_block];
            }
            previous_node = tmp_nodes[cur_row_block];

            // Identify row blocks with only diagonal elements
            if (only_diag)
            {
                ret->start_nodes.push_back(tmp_nodes[cur_row_block]);
            }
        }
        else
        {
            //printf("warp_level\n");
            int cur_row_block;
            for (int i = row_st; i < row_ed; i++)
            {
                hash_idx[i] = ret->global_node;

                cur_row_block = ret->global_node;

                tmp_nodes[cur_row_block] = new node(ret->global_node,
                i, i + 1, csrRowPtr[i + 1] - csrRowPtr[i]);
                ret->global_node++;

                vector<int> dep_list;

                int only_diag = 1;
                for (int idx = csrRowPtr[i]; idx < csrRowPtr[i+1]; idx++)
                {
                    int col_idx = csrColIdx[idx];
                    int dep_row_block = hash_idx[col_idx];
                    if (dep_row_block < cur_row_block)
                    {
                        dep_list.push_back(dep_row_block);
                        only_diag = 0;
                    }
                }

                sort(dep_list.begin(), dep_list.end());
                auto end_pos = unique(dep_list.begin(), dep_list.end());

                for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
                {
                    int dep_row_block = *(dep_i);

                    tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
                    tmp_nodes[dep_row_block]->out_degree++;
                    tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
                    tmp_nodes[cur_row_block]->in_degree++;
                    if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
                        tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
                    ret->global_edge++;
                }

                if (previous_node)
                {
                    previous_node->locality_node = tmp_nodes[cur_row_block];
                }
                previous_node = tmp_nodes[cur_row_block];

                // Identify row blocks with only diagonal elements
                if (only_diag)
                {
                    //printf("only diag node %d\n", cur_row_block);
                    ret->start_nodes.push_back(tmp_nodes[cur_row_block]);
                }

            }
        }
    }

    return ret;
}

ptr_graph generate_graph_row_block_supernode(const int m, const int nnz,
                const int *csrRowPtr, const int *csrColIdx, int thresh)
{
    ptr_graph ret = new graph();

    ptr_node tmp_nodes[m];

    int hash_idx[m];

    for (int i = 0; i < m; i++) hash_idx[i] = -1;

    ptr_node previous_node = NULL;

    for (int row_st = 0; row_st < m;)
    {
        int row_ed = row_st + 1;
        int flag = 1;
        while (flag && row_ed < m && row_ed < row_st + WARP_SIZE)
        {
            // diagonal element do not count
            int rp1 = csrRowPtr[row_st];
            int rp1_ed = csrRowPtr[row_st + 1] - 1;
            int rp2 = csrRowPtr[row_ed];
            int rp2_ed = csrRowPtr[row_ed + 1] - 1;

            int correct = 1, total = 1;
            while (rp1 < rp1_ed || (rp2 < rp2_ed && csrColIdx[rp2] < row_st))
            {
                total++;
                if (csrColIdx[rp1] == csrColIdx[rp2])
                {
                    rp1++;
                    rp2++;
                    correct++;
                }
                else if (csrColIdx[rp1] > csrColIdx[rp2]) rp1++;
                else rp2++;
            }
            float correct_rate = correct / total;

            if (correct_rate == 1.0) row_ed++; else flag = 0;
        }

        //if (row_ed > row_st + 1) printf("supernode %d %d\n", row_st, row_ed);

        for (int i = row_st; i < row_ed; i++)
            hash_idx[i] = ret->global_node;

        int cur_row_block = ret->global_node;

        tmp_nodes[cur_row_block] = new node(ret->global_node,
        row_st, row_ed, csrRowPtr[row_ed] - csrRowPtr[row_st]);
        
        ret->global_node++;

        vector<int> dep_list;

        int only_diag = 1;
        for (int idx = csrRowPtr[row_st]; idx < csrRowPtr[row_ed]; idx++)
        {
            int col_idx = csrColIdx[idx];
            int dep_row_block = hash_idx[col_idx];
            if (dep_row_block < cur_row_block)
            {
                dep_list.push_back(dep_row_block);
                only_diag = 0;
            }
        }

        sort(dep_list.begin(), dep_list.end());
        auto end_pos = unique(dep_list.begin(), dep_list.end());

        for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
        {
            int dep_row_block = *(dep_i);

            tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
            tmp_nodes[dep_row_block]->out_degree++;
            tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
            tmp_nodes[cur_row_block]->in_degree++;
            if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
                tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
            ret->global_edge++;
        }

        if (previous_node)
        {
            previous_node->locality_node = tmp_nodes[cur_row_block];
        }
        previous_node = tmp_nodes[cur_row_block];

        // Identify row blocks with only diagonal elements
        if (only_diag)
        {
            ret->start_nodes.push_back(tmp_nodes[cur_row_block]);
        }

        row_st = row_ed;
    }

    return ret;

}

// ptr_graph generate_graph_row_block_subwarp(const int m, const int nnz,
//                 const int *csrRowPtr, const int *csrColIdx, int row_block, int subwarp_size)
// {
//     ptr_graph ret = new graph();

//     std::vector<ptr_node> node_list;

//     ptr_node previous_node = new node(ret->global_node);
//     vector<int> dep_list;
//     ret->global_node++;
//     int only_diag = 1;

//     int last_level = 0;
//     int current_size = 0;

//     int hash_idx[m];

//     for (int i = 0; i < m; i++) hash_idx[i] = -1;

//     for (int row = 0; row < m; row += row_block)
//     {
//         int row_ed = row + row_block;
//         int cur_row_block = node_list.size();

        
//         for (int idx = csrRowPtr[row]; idx < csrRowPtr[row_ed]; idx++)
//         {
//             int col_idx = csrColIdx[idx];
//             int dep_row_block = has_idx[col_idx];
//             if (dep_row_block < cur_row_block)
//             {
//                 dep_list.push_back(dep_row_block);
//                 only_diag = 0;
//             }
//         }

//         sort(dep_list.begin(), dep_list.end());
//         auto end_pos = unique(dep_list.begin(), dep_list.end());

//         for (auto dep_i = dep_list.begin(); dep_i != end_pos; dep_i++)
//         {
//             int dep_row_block = *(dep_i);

//             tmp_nodes[dep_row_block]->child.push_back(tmp_nodes[cur_row_block]);
//             tmp_nodes[dep_row_block]->out_degree++;
//             tmp_nodes[cur_row_block]->parent.push_back(tmp_nodes[dep_row_block]);
//             tmp_nodes[cur_row_block]->in_degree++;
//             if (tmp_nodes[dep_row_block]->topo_level + 1 > tmp_nodes[cur_row_block]->topo_level)
//                 tmp_nodes[cur_row_block]->topo_level = tmp_nodes[dep_row_block]->topo_level + 1;
//             ret->global_edge++;
//         }

//         if (previous_node)
//         {
//             previous_node->locality_node = tmp_nodes[cur_row_block];
//         }
//         previous_node = tmp_nodes[cur_row_block];

//         // Identify row blocks with only diagonal elements
//         if (only_diag)
//         {
//             ret->start_nodes.push_back(tmp_nodes[cur_row_block]);
//         }
//     }

//     return ret;
// }

void graph_reorder_with_level(ptr_handler handler)
{
    ptr_graph g = handler->graph;

    queue<ptr_node> topo_queue;
    for (auto i = g->start_nodes.begin(); i != g->start_nodes.end(); i++)
    {
        topo_queue.push((*i));
    }

    int row_pos = 0;
    ptr_node last_node = NULL;

    while(!topo_queue.empty())
    {
        ptr_node current_node = topo_queue.front();
        for (auto iter = current_node->child.begin(); iter != current_node->child.end(); iter++)
        {
            (*iter)->in_degree_tmp++;
            if ((*iter)->in_degree_tmp == (*iter)->in_degree)
            {
                topo_queue.push(*iter);
                (*iter)->in_degree_tmp = 0;
            }
        }
        if (last_node) last_node->locality_node = current_node;
        int node_len = current_node->info.end_row - current_node->info.start_row;
        current_node->info.start_row = row_pos;
        current_node->info.end_row = row_pos + node_len;
        row_pos += node_len;
        last_node = current_node;

        topo_queue.pop();
    }
    last_node->locality_node = NULL;
}

void merge_with_size(ptr_handler handler, const int size)
{
    ptr_graph g = handler->graph;

    for (ptr_node i = g->start_nodes[0]; i != NULL;)
    {
        int t_size = 0;
        while (i->locality_node != NULL && t_size < size)
            t_size++;
    }
}

ptr_handler SpTRSV_preprocessing(const int m, const int nnz,
                const int *csrRowPtr, const int *csrColIdx,
                PREPROCESSING_STRATEGY strategy, int row_block)
{
    ptr_handler ret = new SpTRSV_handler();

    ret->m = m;
    ret->nnz = nnz;
    ret->row_block = row_block;

    // printf("???\n");

    if (strategy == ROW_BLOCK)
    {
        ret->graph = generate_graph_row_block(m, nnz, csrRowPtr, csrColIdx, row_block);
    }
    else if (strategy == ROW_BLOCK_NODEP)
    {
        ret->graph = generate_graph_row_block_nodep(m, nnz, csrRowPtr, csrColIdx, row_block);
    }
    else if (strategy == ROW_BLOCK_THRESH)
    {
        ret->graph = generate_graph_row_block_thresh(m, nnz, csrRowPtr, csrColIdx, row_block);
    }
    else if (strategy == ROW_BLOCK_AVG)
    {
        ret->graph = generate_graph_row_block_avg(m, nnz, csrRowPtr, csrColIdx, row_block);
    }
    else if (strategy == SUPERNODE_BLOCK)
    {
        ret->graph = generate_graph_row_block_supernode(m, nnz, csrRowPtr, csrColIdx, row_block);
    }
    else
    {
        // Not implemented
        printf("Error: scheduling strategy not implemented!\n");
    }

    hipMalloc(&ret->get_value, ret->m * sizeof(int));
    hipMemset(ret->get_value, 0, ret->m * sizeof(int));

    hipMalloc(&ret->warp_runtime, ret->m * sizeof(int));
    hipMemset(ret->warp_runtime, 0, ret->m * sizeof(int));

    ret->schedule_level = NULL;
    ret->schedule_info = NULL;
    ret->schedule_subwarp_info = NULL;

    ret->schedule_level_d = NULL;
    ret->schedule_info_d = NULL;
    ret->subwarp_info_d = NULL;

    ret->no_schedule_info = NULL;
    ret->no_schedule_info_d = NULL;

    // for (auto i = ret->graph->start_nodes.begin(); i != ret->graph->start_nodes.end(); i++)
    // {
    //     printf("id %d row %d\n", (*i)->id, (*i)->info.start_row);
    // }

    return ret;
}

void show_graph_layer(ptr_handler handler)
{
    ptr_graph graph = handler->graph;

    int max_layer = -1;
    for (ptr_node i = graph->start_nodes[0]; i != NULL; i = i->locality_node)
    {
        if (i->topo_level > max_layer) max_layer = i->topo_level;
    }

    int layer[max_layer + 1];
    for (int i = 0; i < max_layer; i++)
        layer[i] = 0;

    // printf("Total node number      : %d\n", graph->global_node);
    // printf("Total topological layer: %d\n", max_layer);
    // int count = 0;
    // for (ptr_node i = graph->start_nodes[0]; i != NULL; i = i->locality_node)
    // {
    //     layer[i->topo_level]++;

    //     //printf("node %d layer %d\n", count, i->topo_level);
    //     count++;
    // }
    // printf("layer distribution: ");
    // for (int i = 0; i < max_layer; i++)
    //     printf("%d ", layer[i]);
    // printf("\n");

}

void write_graph(const char* file_name, ptr_handler handler, unsigned int max_depth,
int layer, float parallelism)
{
    string f1 = file_name;
    f1 += ".global";
    string f2 = file_name;
    f2 += ".node";
    string f3 = file_name;
    f3 += ".edge";

    FILE* fp1 = fopen(f1.c_str(), "w");
    FILE* fp2 = fopen(f2.c_str(), "w");
    FILE* fp3 = fopen(f3.c_str(), "w");

    ptr_graph g = handler->graph;

    int node_flag[g->global_node];
    memset(node_flag, 0, g->global_node * sizeof(int));

    queue<ptr_node> node_queue;
    ptr_node depth_ptr = NULL;

    for (auto i = g->start_nodes.begin(); i != g->start_nodes.end(); i++)
    {
        node_queue.push(*i);
        if (i + 1 == g->start_nodes.end())
            depth_ptr = *i;
        node_flag[(*i)->id] = 1;
    }
    
    int num_node = 0;
    int num_edge = 0;
    // Get number of nodes and edges
    int current_depth = 0;
    while (!node_queue.empty() && current_depth <= max_depth)
    {
        ptr_node current_node = node_queue.front();

        if (current_depth < max_depth)
        {
            for (auto i = current_node->child.begin(); i != current_node->child.end(); i++)
            {
                if (current_node->topo_level <= max_depth)
                {
                    if (!node_flag[(*i)->id])
                    {
                        node_queue.push(*i);
                        node_flag[(*i)->id] = 1;
                    }
                    num_edge++;
                    fprintf(fp3, "%d %d\n", current_node->id, (*i)->id);
                }
            }
        }
        if (current_node == depth_ptr)
        {
            current_depth++;
            depth_ptr = node_queue.back();
        }

        // output node information
        // feature: number of nnzs
        fprintf(fp2, "%d %d %d\n", current_node->id,
        current_node->child.size(), current_node->parent.size());
        num_node++;

        // start row id / number of rows,
        // start nnz id / number of nnzs,
        // end nnz id / number of nnzs, 
        // number of row nnzs / number of nnzs

        node_queue.pop();
    }

    // output global information
    fprintf(fp1, "%d %d %d %f\n", num_node, num_edge, layer, parallelism);
}

inline int lowbit(int x)
{
    return x & (-x);
}
void update_levels(int pos, int maxn, int *a)
{
    for (int i = pos; i < maxn; i += lowbit(i))
        a[i]++;
}
int prefixsum_levels(int pos, int *a)
{
    int ans = 0;
    for (int i = pos; i; i -= lowbit(i))
        ans += a[i];
    return ans;
}


// Features to output
// 1. avg_nnz        : Average number of non-zero elements per row
// 2. csrCoefficient : Coefficient of Variance of number of non-zero elements per row
// 3. Parallelism    : Average number of non-zero elements per level
// 4. parCoefficient : Coefficient of Variance of number of non-zero elements per level

void get_matrix_info(const int    m,
                const int         nnz,
                const int        *csrRowPtr,
                const int        *csrColIdx,
                float            *avg_rnnz,
                float            *cov_rnnz,
                float            *avg_lnnz,
                float            *cov_lnnz,
                float            *dep_dist,
                float            *reverse_level)
{
    int total_nnz = 0;

    int *layer=(int*)malloc(m * sizeof(int));
    if (layer == NULL)
        printf("layer error\n");
    memset(layer, 0, sizeof(int) * m);

    int *layer_num = (int*)malloc((m + 1) * sizeof(int));
    if (layer_num == NULL)
        printf("layer_num error\n");
    memset (layer_num, 0, sizeof(int) * (m + 1));

    int max_layer;
    int max_layer2 = 0;
    int max = 0;
    unsigned int min = -1;

    int sum_level[m + 1];
    for (int i = 0; i < m + 1; i++)
    {
        sum_level[i] = 0;
    }

    *reverse_level = 0;

    // count layer
    int row, j;
    for (row = 0; row < m; row++)
    {
        max_layer = 0;

        total_nnz += csrRowPtr[row+1] - csrRowPtr[row];

        for (j = csrRowPtr[row]; j < csrRowPtr[row+1]; j++)
        {
            int col = csrColIdx[j];

            if((layer[col] + 1) > max_layer)
                max_layer = layer[col] + 1;

        }
        layer[row] = max_layer;

        float reverse_item = row - prefixsum_levels(max_layer, sum_level);
        //printf("row %d reverse_item %f\n", row, reverse_item);
        reverse_item = reverse_item / (row + 1);
        *reverse_level = *reverse_level + reverse_item;
        
        update_levels(max_layer, m + 1, sum_level);

        layer_num[max_layer]++;
        if (max_layer > max_layer2)
            max_layer2 = max_layer;
    }

    *reverse_level = *reverse_level / m;

    *avg_rnnz = 1.0 * total_nnz / m;

    *dep_dist = 0;

    *cov_rnnz = 0;
    for (int i = 0; i < m; i++)
    {
        int row_nnz = csrRowPtr[i+1] - csrRowPtr[i];
        *cov_rnnz += (row_nnz - *avg_rnnz) * (row_nnz - *avg_rnnz);
        if (csrRowPtr[i+1] - csrRowPtr[i] > 1)
            *dep_dist = *dep_dist + 1.0 / (i - csrColIdx[csrRowPtr[i + 1] - 2]);
    }
    *cov_rnnz = sqrt(*cov_rnnz / m) / *avg_rnnz;
    *dep_dist = *dep_dist / m;

    int total_layer_num = 0;
    for(j = 1; j <= max_layer2; j++)
    {
        if(max < layer_num[j])
            max = layer_num[j];
        if(min > layer_num[j])
            min = layer_num[j];
        total_layer_num += layer_num[j];
    }
    *avg_lnnz = 1.0 * total_layer_num / max_layer2;

    *cov_lnnz = 0;
    for(j = 1; j <= max_layer2; j++)
    {
        *cov_lnnz += (layer_num[j] - *avg_lnnz) * (layer_num[j] - *avg_lnnz);
    }
    *cov_lnnz = sqrt(*cov_lnnz / max_layer2) / *avg_lnnz;

    free(layer);
    free(layer_num);
}

void write_matrix_info(const char* file_name,
                const char*       matrix_name,
                const int         m,
                const int         nnz,
                const int        *csrRowPtr,
                const int        *csrColIdx)
{
    float avg_rnnz, cov_rnnz, avg_lnnz, cov_lnnz, dep_dist, reverse_level;
    get_matrix_info(m, nnz, csrRowPtr, csrColIdx,
    &avg_rnnz, &cov_rnnz, &avg_lnnz, &cov_lnnz, &dep_dist, &reverse_level);

    FILE *fp = fopen(file_name, "a");
    fprintf(fp, "%s,%d,%d,%.2f,%.2f,%.2f,%.2f,%.2f,%.2f\n", matrix_name,
    m, nnz, avg_rnnz, cov_rnnz, avg_lnnz, cov_lnnz, dep_dist, reverse_level);
    fclose(fp);
    
}

void get_matrix_info2(const int    m,
                const int         nnz,
                const int        *csrRowPtr,
                const int        *csrColIdx,
                float *out_args,
                int size)
{
    if (size != 11)
    {
        printf("size must be 11!\n");
        return;
    }

    int *layer=(int*)malloc(m * sizeof(int));
    if (layer == NULL)
        printf("layer error\n");
    memset(layer, 0, sizeof(int) * m);

    int *layer_num = (int*)malloc((m + 1) * sizeof(int));
    if (layer_num == NULL)
        printf("layer_num error\n");
    memset (layer_num, 0, sizeof(int) * (m + 1));

    int total_layers = 0;

    int sum_level[m + 1];
    for (int i = 0; i < m + 1; i++)
    {
        sum_level[i] = 0;
    }

    float reverse_level = 0;

    int max_rnnz = -1;

    // count layer
    int row, j;
    for (row = 0; row < m; row++)
    {
        int max_layer = 0;

        if (max_rnnz < csrRowPtr[row+1] - csrRowPtr[row])
        {
            max_rnnz = csrRowPtr[row+1] - csrRowPtr[row];
        }

        for (j = csrRowPtr[row]; j < csrRowPtr[row+1]; j++)
        {
            int col = csrColIdx[j];

            if((layer[col] + 1) > max_layer)
                max_layer = layer[col] + 1;

        }
        layer[row] = max_layer;

        // float reverse_item = row - prefixsum_levels(max_layer, sum_level);
        // //printf("row %d reverse_item %f\n", row, reverse_item);
        // reverse_item = reverse_item / (row + 1);
        // reverse_level = reverse_level + reverse_item;
        
        // update_levels(max_layer, m + 1, sum_level);

        layer_num[max_layer]++;
        if (max_layer > total_layers)
            total_layers = max_layer;
    }

    reverse_level = reverse_level / m;

    float avg_rnnz = 1.0 * nnz / m;

    float dep_dist = 0;

    float cov_rnnz = 0;
    for (int i = 0; i < m; i++)
    {
        int row_nnz = csrRowPtr[i+1] - csrRowPtr[i];
        cov_rnnz += (row_nnz - avg_rnnz) * (row_nnz - avg_rnnz);
        if (csrRowPtr[i+1] - csrRowPtr[i] > 1)
            dep_dist = dep_dist + 1.0 / (i - csrColIdx[csrRowPtr[i + 1] - 2]);
        //printf("%d ", csrColIdx[csrRowPtr[i + 1] - 1] - csrColIdx[csrRowPtr[i + 1] - 2]);
    }
    //printf("dist %.3f\n", dep_dist);
    cov_rnnz = sqrt(cov_rnnz / m) / avg_rnnz;
    dep_dist = dep_dist / m;

    int total_layer_num = 0;
    int max_lnnz = 0;
    unsigned int min_lnnz = -1;
    for(j = 1; j <= total_layers; j++)
    {
        if(max_lnnz < layer_num[j])
            max_lnnz = layer_num[j];
        if(min_lnnz > layer_num[j])
            min_lnnz = layer_num[j];
        total_layer_num += layer_num[j];
    }
    float avg_lnnz = 1.0 * total_layer_num / total_layers;

    float cov_lnnz = 0;
    for(j = 1; j <= total_layers; j++)
    {
        cov_lnnz += (layer_num[j] - avg_lnnz) * (layer_num[j] - avg_lnnz);
    }
    cov_lnnz = sqrt(cov_lnnz / total_layers) / avg_lnnz;

    int idx = 0;
    out_args[idx++] = m;
    out_args[idx++] = nnz;
    out_args[idx++] = avg_rnnz;
    out_args[idx++] = max_rnnz;
    out_args[idx++] = cov_rnnz;
    out_args[idx++] = avg_lnnz;
    out_args[idx++] = max_lnnz;
    out_args[idx++] = cov_lnnz;
    out_args[idx++] = dep_dist;
    out_args[idx++] = total_layers;
    out_args[idx++] = reverse_level;

    free(layer);
    free(layer_num);
}

void write_matrix_info2(const char* file_name,
                const char*       matrix_name,
                const int         m,
                const int         nnz,
                const int        *csrRowPtr,
                const int        *csrColIdx)
{
    const int size = 11;
    float out_args[size];
    //float avg_rnnz, cov_rnnz, avg_lnnz, cov_lnnz, dep_dist, reverse_level;
    get_matrix_info2(m, nnz, csrRowPtr, csrColIdx, out_args, size);

    FILE *fp = fopen(file_name, "a");
    fprintf(fp, "%s,", matrix_name);
    for (int i = 0; i < size - 1; i++)
        fprintf(fp, "%.2f,", out_args[i]);
    fprintf(fp, "%.2f\n", out_args[size - 1]);
    fclose(fp);
}